#include "hip/hip_runtime.h"
#ifndef UTIL_CU_
#define UTIL_CU_

// TITAN V : SM:80
// GTX 1080: smem 40x1024, gmem 80x512
// GTX 980: 32x1024
// K40: 30x1024
// TRX 2080: smem 92x1024
#ifndef NUM_BLOCKS
#define NUM_BLOCKS 80
#endif

#define NUM_THREADS 1024

#define MAX_RK_SIZE 182

__device__ void load_roundkey(uint *s, uint *rk) {
    int tid = threadIdx.x;
    if (tid < MAX_RK_SIZE)
	s[tid] = rk[tid];
    __syncthreads();
}

__device__ void load_smem_sbox(uchar *smem, uchar *gmem) {
    int tid = threadIdx.x;
    if (tid < 256)
	smem[tid] = gmem[tid];
    __syncthreads();
}

#if NUM_THREADS == 1024
__device__ void load_smem(uchar *st0, uchar *gt0, uchar *st1, uchar *gt1, uchar *st2, uchar *gt2, uchar *st3, uchar *gt3) {
    int tid = threadIdx.x;
    uint *s, *g;
    if (tid < 256) {
	s = (uint *)st0; g = (uint *)gt0;
	s[tid] = g[tid];
    } else if (tid < 512) {
	tid -= 256;
	s = (uint *)st1; g = (uint *)gt1;
	s[tid] = g[tid];
    } else if (tid < 768) {
	tid -= 512;
	s = (uint *)st2; g = (uint *)gt2;
	s[tid] = g[tid];
    } else {
	tid -= 768;
	s = (uint *)st3; g = (uint *)gt3;
	s[tid] = g[tid];
    }
    
    __syncthreads();
}
__device__ void load_smem(uint *st0, uint *gt0, uint *st1, uint *gt1, uint *st2, uint *gt2, uint *st3, uint *gt3) {
    int tid = threadIdx.x;
    if (tid < 256) {
	st0[tid] = gt0[tid];
    } else if (tid < 512) {
	tid -= 256;
	st1[tid] = gt1[tid];
    } else if (tid < 768) {
	tid -= 512;
	st2[tid] = gt2[tid];
    } else {
	tid -= 768;
	st3[tid] = gt3[tid];
    }
    
    __syncthreads();
}
#elif NUM_THREADS == 512
__device__ void load_smem(uchar *st0, uchar *gt0, uchar *st1, uchar *gt1, uchar *st2, uchar *gt2, uchar *st3, uchar *gt3) {
    int tid = threadIdx.x;
    uint *s, *g;
    if (tid < 256) {
	s = (uint *)st0; g = (uint *)gt0;
	s[tid] = g[tid];
	s = (uint *)st2; g = (uint *)gt2;
	s[tid] = g[tid];
    } else {
	tid -= 256;
	s = (uint *)st1; g = (uint *)gt1;
	s[tid] = g[tid];
	s = (uint *)st3; g = (uint *)gt3;
	s[tid] = g[tid];
    }
    
    __syncthreads();
}
__device__ void load_smem(uint *st0, uint *gt0, uint *st1, uint *gt1, uint *st2, uint *gt2, uint *st3, uint *gt3) {
    int tid = threadIdx.x;
    if (tid < 256) {
	st0[tid] = gt0[tid];
	st2[tid] = gt2[tid];
    } else {
	tid -= 256;
	st1[tid] = gt1[tid];
	st3[tid] = gt3[tid];
    }
    
    __syncthreads();
}

#elif NUM_THREADS == 256
__device__ void load_smem(uchar *st0, uchar *gt0, uchar *st1, uchar *gt1, uchar *st2, uchar *gt2, uchar *st3, uchar *gt3) {
    int tid = threadIdx.x;
    uint *s, *g;
    s = (uint *)st0; g = (uint *)gt0;
    s[tid] = g[tid];
    s = (uint *)st1; g = (uint *)gt1;
    s[tid] = g[tid];
    s = (uint *)st2; g = (uint *)gt2;
    s[tid] = g[tid];
    s = (uint *)st3; g = (uint *)gt3;
    s[tid] = g[tid];
    
    __syncthreads();
}
__device__ void load_smem(uint *st0, uint *gt0, uint *st1, uint *gt1, uint *st2, uint *gt2, uint *st3, uint *gt3) {
    int tid = threadIdx.x;
    st0[tid] = gt0[tid];
    st1[tid] = gt1[tid];
    st2[tid] = gt2[tid];
    st3[tid] = gt3[tid];
    
    __syncthreads();
}
#else 
#error NUM_THREADS must be 256, 512 or 1024
#endif // NUM_THREADS

#endif

